#include "hip/hip_runtime.h"
#include "FFT.h"
#include "logger.h"
#include "parser.h"
#include <cmath>
#include <crt/host_defines.h>
#include <cstdlib>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <math.h>
#include <iostream>
#include <string>
#include <vector>
#include <algorithm>
#include <chrono>
#include <fstream>

using namespace std;

template <typename T>
ostream &operator<<(ostream &o, vector<T> v) {
    if (v.size() > 0) {
        o << v[0];
    }
    for (unsigned i = 1; i < v.size(); i++) {
        o << " " << v[i];
    }

    return o << endl;
}

static __device__ __host__ inline cmplx_struct add(cmplx_struct a, cmplx_struct b) {
    cmplx_struct num;
    num.x = a.x + b.x;
    num.y = a.y + b.y;
    return num;
}

static __device__ __host__ inline cmplx_struct inverse(cmplx_struct number) {
    cmplx_struct inverse;
    inverse.x = -number.x;
    inverse.y = -number.y;
    return inverse;
}

static __device__ __host__ inline cmplx_struct multiply(cmplx_struct first, cmplx_struct second) {
    cmplx_struct mult;
    mult.x = first.x * second.x - first.y * second.y;
    mult.y = first.y * second.x + first.x * second.y;
    return mult;
}

__global__ void inverse_divide(cmplx_struct* numbers, int n, int threads) {
    int index = blockIdx.x * threads + threadIdx.x;
    // bounds check
    if (index < n) {
        numbers[index].x /= n;
        numbers[index].y /= n;
    } 
}

// TODO pointer aliasing optimization must go in the report
// https://developer.nvidia.com/blog/cuda-pro-tip-optimize-pointer-aliasing/
__global__ void reorder_array(cmplx_struct* __restrict__ rev, cmplx_struct* __restrict__ orig, int s, int threads, int n) {
    unsigned int index = blockIdx.x * threads + threadIdx.x;
    if (index < n && (__brev(index) >> (32 - s)) < n) {
        // reversed
        rev[__brev(index) >> (32 - s)] = orig[index];
    }
}

__device__ void fft_inner_loop(cmplx_struct* __restrict__ numbers, int row, int col, int len, int n, bool invert) {
    if (row + col + len / 2 < n && col < len / 2) {
        cmplx_struct first, second;
        float angle = (ANGLE_MULT * col) / (len * (invert ? 1 : -1));
        second.x = cos(angle);
        second.y = sin(angle);

        first = numbers[row + col];
        second = multiply(numbers[row + col + len / 2], second);

        numbers[row + col] = add(first, second);
        numbers[row + col + len / 2] = add(first, inverse(second));
    }
}


__global__ void compute_fft(cmplx_struct* __restrict__ numbers, int row, int len, int n, int threads, bool invert) {
    int col = blockIdx.x * threads + threadIdx.x;
    fft_inner_loop(numbers, row, col, len, n, invert);
}

__global__ void fft_outer_loop(cmplx_struct* __restrict__ numbers, int len, int n, int threads, bool invert) {
    int row = (blockIdx.x * threads + threadIdx.x) * len;

    for (int col = 0; col < len / 2; col++) {
        fft_inner_loop(numbers, row, col, len, n, invert);
    }
}

void real_fft(int size, int threads, cmplx_struct* reversed_nums, cmplx_struct* nums, int balance, bool invert) {
    int power = log2(size);

    auto start = chrono::high_resolution_clock::now();

    reorder_array<<<ceil(float(size) / threads), threads>>>(reversed_nums, nums, power, threads, size);

    // need to wait for all the bits to be reversed
    hipDeviceSynchronize();

    // parallel fft
    for (int len = 2; len <= size; len <<= 1) {
        if (size / len > balance) {
            fft_outer_loop<<<ceil((float) size / threads / len), threads>>>(reversed_nums, len, size, threads, invert);
        } else {
            for (int row = 0; row < size; row += len) {
                float repeat = (float) len / 2;
                compute_fft<<<ceil(repeat / threads), threads>>>(reversed_nums, row, len, size, threads, invert);
            }
        }
    }

    if (invert) {
        inverse_divide<<<ceil((float) size / threads), threads>>>(reversed_nums, size, threads);
    }

    auto stop = chrono::high_resolution_clock::now();
    auto duration = chrono::duration_cast<chrono::microseconds>(stop - start);
    cout << duration.count() << endl;

}


void fft(vector<cmplx>& array, bool invert, int balance, int threads) {
    int size = (int) array.size();
    cmplx_struct* data = (cmplx_struct*)malloc(sizeof(cmplx_struct) * size);
    for (int i = 0; i < size; i++) {
        data[i].x = array[i].real();
        data[i].y = array[i].imag();
    }

    cmplx_struct *reversed_nums, *nums;
    hipMalloc((void **)&reversed_nums, sizeof(cmplx_struct) * size);
    hipMalloc((void **)&nums, sizeof(cmplx_struct) * size);
    hipMemcpy(nums, data, sizeof(cmplx_struct) * size, hipMemcpyHostToDevice);

    real_fft(size, threads, reversed_nums, nums, balance, invert);

    cmplx_struct* results;
    results = (cmplx_struct*)malloc(sizeof(cmplx_struct) * size);
    hipMemcpy(results, reversed_nums, sizeof(cmplx_struct) * size, hipMemcpyDeviceToHost);

    for (int i = 0; i < size; i++) {
        array[i] = cmplx(results[i].x, results[i].y);
    }

    // cleanup
    free(data);
    hipFree(reversed_nums);
    hipFree(nums);
}

void fft_2D(vector<vector<cmplx> >& data, bool invert, int thread_balance, int threads) {
    vector<vector<cmplx> >& matrix = data;
    for (int i = 0; i < matrix.size(); i++) {
        fft(matrix[i], invert, thread_balance, threads);
    }

    data = matrix;
    matrix.resize(data[0].size());
    for (int i = 0; i < matrix.size(); i++) {
        matrix[i].resize(data.size());
    }

    for (int i = 0; i < data.size(); i++) {
        for (int j = 0; j < data[0].size(); j++) {
            matrix[j][i] = data[i][j];
        }
    }

    for (int i = 0; i < matrix.size(); i++) {
        fft(matrix[i], invert, thread_balance, threads);
    }

    for (int i = 0; i < data.size(); i++) {
        for (int j = 0; j < data[0].size(); j++) {
            data[j][i] = matrix[i][j];
        }
    }

}


vector<int> multiply_poly(vector<int> first, vector<int> second, int thread_balance, int threads) {
    vector<cmplx> cmplx_poly1(first.begin(), first.end()), cmplx_poly2(second.begin(), second.end());

    int size = 1;
    int maximum = max(first.size(), second.size());
    while (size < maximum) {
        size <<= 1;
    }
    size <<= 1;

    cmplx_poly1.resize(size);
    cmplx_poly2.resize(size);

    fft(cmplx_poly1, false, thread_balance, threads);
    fft(cmplx_poly2, false, thread_balance, threads);

    for (int i = 0; i < size; i++) {
        cmplx_poly1[i] *= cmplx_poly2[i];
    }

    fft(cmplx_poly1, true, thread_balance, threads);

    vector<int> results;
    results.resize(size);
    for (int i = 0; i < size; i++) {
        results[i] = int(cmplx_poly1[i].real() + 0.5);
    }

    return results;
}


int main(int argc, char** argv) {

    Parser* ip = new Parser();
    ip->parse(argc, argv);
    vector<cmplx> result(ip->length);
    generate(result.begin(), result.end(), rand);

    auto start = chrono::high_resolution_clock::now();
    fft(result, false, 2, ip->threads);
    auto stop = chrono::high_resolution_clock::now();
    auto duration = chrono::duration_cast<chrono::microseconds>(stop - start);
    // cout << duration.count() << endl;

    // vector<int> result = multiply_poly(fa, fb, 2, 1024);
    // for (int i = 0; i < result.size(); i++) {
    //     cout << result[i] << " ";
    // }
    // cout << "\n";
    
}

