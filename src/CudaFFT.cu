#include "hip/hip_runtime.h"
#include "FFT.h"
#include "logger.h"
#include "parser.h"
#include <cmath>
#include <crt/host_defines.h>
#include <cstdlib>
#include <cuda_device_runtime_api.h>
#include <hip/hip_runtime_api.h>
#include <hip/driver_types.h>
#include <math.h>
#include <iostream>
#include <string>
#include <vector>
#include <algorithm>
#include <chrono>
#include <fstream>
// #include <opencv2/core/core.hpp>
// #include <opencv2/highgui/highgui.hpp>

using namespace std;

template <typename T>
ostream &operator<<(ostream &o, vector<T> v) {
    if (v.size() > 0) {
        o << v[0];
    }
    for (unsigned i = 1; i < v.size(); i++) {
        o << " " << v[i];
    }

    return o << endl;
}

static __device__ __host__ inline cmplx_struct add(cmplx_struct a, cmplx_struct b) {
    cmplx_struct num;
    num.x = a.x + b.x;
    num.y = a.y + b.y;
    return num;
}

static __device__ __host__ inline cmplx_struct inverse(cmplx_struct number) {
    cmplx_struct inverse;
    inverse.x = -number.x;
    inverse.y = -number.y;
    return inverse;
}

static __device__ __host__ inline cmplx_struct multiply(cmplx_struct first, cmplx_struct second) {
    cmplx_struct mult;
    mult.x = first.x * second.x - first.y * second.y;
    mult.y = first.y * second.x + first.x * second.y;
    return mult;
}

__global__ void inverse_divide(cmplx_struct* numbers, int n, int threads) {
    int index = blockIdx.x * threads + threadIdx.x;
    // bounds check
    if (index < n) {
        numbers[index].x /= n;
        numbers[index].y /= n;
    } 
}

// TODO pointer aliasing optimization must go in the report
// https://developer.nvidia.com/blog/cuda-pro-tip-optimize-pointer-aliasing/
__global__ void reorder_array(cmplx_struct* __restrict__ rev, cmplx_struct* __restrict__ orig, int s, int threads, int n) {
    unsigned int index = blockIdx.x * threads + threadIdx.x;
    if (index < n && (__brev(index) >> (32 - s)) < n) {
        // reversed
        rev[__brev(index) >> (32 - s)] = orig[index];
    }
}

__device__ void fft_inner_loop(cmplx_struct* __restrict__ numbers, int row, int col, int len, int n, bool invert) {
    if (row + col + len / 2 < n && col < len / 2) {
        cmplx_struct first, second;
        float angle = (ANGLE_MULT * col) / (len * (invert ? 1 : -1));
        second.x = cos(angle);
        second.y = sin(angle);

        first = numbers[row + col];
        second = multiply(numbers[row + col + len / 2], second);

        numbers[row + col] = add(first, second);
        numbers[row + col + len / 2] = add(first, inverse(second));
    }
}


__global__ void compute_fft(cmplx_struct* __restrict__ numbers, int row, int len, int n, int threads, bool invert) {
    int col = blockIdx.x * threads + threadIdx.x;
    fft_inner_loop(numbers, row, col, len, n, invert);
}

__global__ void fft_outer_loop(cmplx_struct* __restrict__ numbers, int len, int n, int threads, bool invert) {
    int row = (blockIdx.x * threads + threadIdx.x) * len;

    for (int col = 0; col < len / 2; col++) {
        fft_inner_loop(numbers, row, col, len, n, invert);
    }
}

void real_fft(int size, int threads, cmplx_struct* reversed_nums, cmplx_struct* nums, int balance, bool invert) {
    int power = log2(size);

    auto start = chrono::high_resolution_clock::now();

    reorder_array<<<ceil(float(size) / threads), threads>>>(reversed_nums, nums, power, threads, size);

    // no need to wait for all the bits to be reversed (implicit sync)
    // hipDeviceSynchronize();

    // parallel fft
    for (int len = 2; len <= size; len <<= 1) {
        if (size / len > balance) {
            fft_outer_loop<<<ceil((float) size / threads / len), threads>>>(reversed_nums, len, size, threads, invert);
        } else {
            for (int row = 0; row < size; row += len) {
                float repeat = (float) len / 2;
                compute_fft<<<ceil(repeat / threads), threads>>>(reversed_nums, row, len, size, threads, invert);
            }
        }
    }

    if (invert) {
        inverse_divide<<<ceil((float) size / threads), threads>>>(reversed_nums, size, threads);
    }

    auto stop = chrono::high_resolution_clock::now();
    auto duration = chrono::duration_cast<chrono::microseconds>(stop - start);
    cout << duration.count() << endl;

}


void fft(vector<cmplx>& array, bool invert, int balance, int threads) {
    int size = (int) array.size();
    cmplx_struct* data = (cmplx_struct*)malloc(sizeof(cmplx_struct) * size);
    for (int i = 0; i < size; i++) {
        data[i].x = array[i].real();
        data[i].y = array[i].imag();
    }

    cmplx_struct *reversed_nums, *nums;
    hipMalloc((void **)&reversed_nums, sizeof(cmplx_struct) * size);
    hipMalloc((void **)&nums, sizeof(cmplx_struct) * size);
    hipMemcpy(nums, data, sizeof(cmplx_struct) * size, hipMemcpyHostToDevice);

    real_fft(size, threads, reversed_nums, nums, balance, invert);

    cmplx_struct* results;
    results = (cmplx_struct*)malloc(sizeof(cmplx_struct) * size);
    hipMemcpy(results, reversed_nums, sizeof(cmplx_struct) * size, hipMemcpyDeviceToHost);

    for (int i = 0; i < size; i++) {
        array[i] = cmplx(results[i].x, results[i].y);
    }

    // cleanup
    free(data);
    hipFree(reversed_nums);
    hipFree(nums);
}

void fft_2D(vector<vector<cmplx> >& data, bool invert, int thread_balance, int threads) {
    vector<vector<cmplx> >& matrix = data;
    for (int i = 0; i < matrix.size(); i++) {
        fft(matrix[i], invert, thread_balance, threads);
    }

    data = matrix;
    matrix.resize(data[0].size());
    for (int i = 0; i < matrix.size(); i++) {
        matrix[i].resize(data.size());
    }

    for (int i = 0; i < data.size(); i++) {
        for (int j = 0; j < data[0].size(); j++) {
            matrix[j][i] = data[i][j];
        }
    }

    for (int i = 0; i < matrix.size(); i++) {
        fft(matrix[i], invert, thread_balance, threads);
    }

    for (int i = 0; i < data.size(); i++) {
        for (int j = 0; j < data[0].size(); j++) {
            data[j][i] = matrix[i][j];
        }
    }

}


void compress_image(vector<vector<uint8_t>> &image, double threshold, int balance, int threads) {
    //Convert image to complex type
    vector<vector<cmplx>> complex_image(image.size(), vector<base>(image[0].size()));
    for (int i = 0; i < image.size(); i++) {
        for (int j = 0; j < image[0].size(); j++) {
            complex_image[i][j] = image[i][j];
        }
    }

    //Perform 2D fft on image
    fft_2D(complex_image, false, balance, threads);

    //Threshold the fft

    double maximum_value = 0.0;
    for (int i = 0; i < complex_image.size(); i++) {
        for (int j = 0; j < complex_image[0].size(); j++) {
            maximum_value = max(maximum_value, abs(complex_image[i][j]));
        }
    }
    threshold *= maximum_value;

    for (int i = 0; i < complex_image.size(); i++) {
        for (int j = 0; j < complex_image[0].size(); j++) {
            if (abs(complex_image[i][j]) < threshold) {
                complex_image[i][j] = 0;
            }
        }
    }
    int zeros_count = 0;
    for (int i = 0; i < complex_image.size(); i++) {
        for (int j = 0; j < complex_image[0].size(); j++) {
            if (abs(complex_image[i][j]) == 0) {
                zeros_count++;
            }
        }
    }
    cout << "Components removed: " << ((zeros_count*1.00/(complex_image.size()*complex_image[0].size())))*100 << endl;

    // Perform inverse FFT
    fft_2D(complex_image, true, balance, threads);

    // We will consider only the real part of the image
    for (int i = 0; i < complex_image.size(); i++) {
        for (int j = 0; j < complex_image[0].size(); j++) {
            image[i][j] = uint8_t(complex_image[i][j].real() + 0.5);
        }
    }

}


int main(int argc, char** argv) {

    // Parser* ip = new Parser();
    // ip->parse(argc, argv);
    // vector<cmplx> result(ip->length);
    // generate(result.begin(), result.end(), rand);

    // auto start = chrono::high_resolution_clock::now();
    vector<cmplx> result = {1, 2, 3, 4, 5, 6, 7, 8};
    fft(result, false, 2, 4);
    // auto stop = chrono::high_resolution_clock::now();
    // auto duration = chrono::duration_cast<chrono::microseconds>(stop - start);
    // cout << duration.count() << endl;

    // vector<int> result = multiply_poly(fa, fb, 2, 1024);
    for (int i = 0; i < result.size(); i++) {
        cout << result[i] << " ";
    }
    cout << "\n";

    // std::vector<int> fa(ip->length);
    // std::generate(fa.begin(), fa.end(), std::rand);
    // std::vector<int> fb(ip->length);
    // std::generate(fb.begin(), fb.end(), std::rand);

    // auto start = chrono::high_resolution_clock::now(); 
    // auto result_parallel = multiply_poly(fa, fb, 4, ip->threads);
    // auto stop = chrono::high_resolution_clock::now();
    // auto duration = chrono::duration_cast<chrono::microseconds>(stop - start);
    // cout << duration.count() << endl;


    
}

